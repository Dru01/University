﻿#include <stdio.h>
#include <stdlib.h> 
#include <omp.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define ll long long

__global__ void Solve(long double *v, long double *v_1, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N) {
		v_1[idx] = v[idx] + v[idx + 1];
	}
}

int divUp(int a, int b) {
	return (a + b - 1) / b;
}

int main() {

	hipSetDevice(0);

	ll N = (ll)1e7;

	long double* v = (long double*)malloc(N * sizeof(long double));
	long double* v_1 = (long double*)malloc((N - 1) * sizeof(long double));

	for (int i = 0; i < N; i++)
		v[i] = (long double)i / exp(1);

	long double* v_device,* v_1_device;
	hipMalloc((void**)&v_device, N * sizeof(long double));
	hipMalloc((void**)&v_1_device, (N-1) * sizeof(long double));
	hipMemcpy(v_device, v, N * sizeof(long double), hipMemcpyHostToDevice);

	clock_t t;
	t = clock();

	int threads = 512, grid = divUp(N, threads);
	Solve <<< grid, threads >>> (v_device, v_1_device, N);
	hipDeviceSynchronize();
	
	t = clock() - t;

	hipFree(v_device);
	hipFree(v_1_device);
	free(v);
	free(v_1);

	printf("%f", ((long double)t) / CLOCKS_PER_SEC);

	return 0;
}