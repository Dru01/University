﻿#include <stdio.h>
#include <stdlib.h> 
#include <omp.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define ll long long

__global__ void Solve(double *A, double *B, double *C, double alpha, int N, int M) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int index = (idy)*M + (idx);
	int index1 = (N - 1 - idy) * M + (M - 1 - idx);
	if (index < N*M) {
		C[index] = (alpha)*A[index] + (1-alpha)*B[index1];
	}
}

int divUp(int a, int b) {
	return (a + b - 1) / b;
}

int main() {

	hipSetDevice(0);

	ll N = (ll)1e4, M = (ll)1e4;

	double* A = (double*)malloc(N * M * sizeof(double)), * B = (double*)malloc(N * M * sizeof(double)), * C = (double*)malloc(N * M * sizeof(double));
	double alpha = 0.5;

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			A[i * M + j] = 1.0;
			B[i * M + j] = i * M + j;
		}
	}

	double* A_device, * B_device, * C_device;
	hipMalloc((void**)&A_device, N * M * sizeof(double));
	hipMalloc((void**)&B_device, N * M * sizeof(double));
	hipMalloc((void**)&C_device, N * M * sizeof(double));
	hipMemcpy(A_device, A, N * M * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(B_device, B, N * M * sizeof(double), hipMemcpyHostToDevice);

	clock_t t;
	t = clock();

	dim3 threads(16, 16, 1), grid(divUp(M, 16), divUp(N, 16), 1);
	Solve <<< grid, threads >>> (A_device, B_device, C_device, alpha, N, M);
	hipDeviceSynchronize();
	
	t = clock() - t;

	hipMemcpy(C, C_device, N * M* sizeof(double), hipMemcpyDeviceToHost);

	hipFree(A_device);
	hipFree(B_device);
	hipFree(C_device);
	free(A);
	free(B);
	free(C);

	printf("%f", ((double)t) / CLOCKS_PER_SEC);

	return 0;
}