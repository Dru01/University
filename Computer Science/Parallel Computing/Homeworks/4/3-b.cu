#include <stdio.h>
#include <stdlib.h> 
#include <omp.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define ll long long

__global__ void mat_prod(double *A, double *B, double *C, int N, int K, int M)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int index = (idy)*M + (idx);
    C[index] = 0.0;
    for (int k = 0; k < K; k++)
        C[index] += A[idy * K + k] * B[idx * K + k];
}

int divUp(int a, int b) {
    return (a + b - 1) / b;
}

int main()
{
    ll N = 1e4, K = 1e3, M = 1e3;

    double* A = (double*)malloc(N * K * sizeof(double)), * B = (double*)malloc(M * K * sizeof(double)), * C = (double*)malloc(N * M * sizeof(double));
    for (int i = 0; i < N; i++)
        for (int j = 0; j < K; j++)
            A[i * K + j] = i * K + j;

    for (int i = 0; i < M; i++)
        for (int j = 0; j < K; j++)
            B[i * K + j] = i * K + j;

    double* A_device, * B_device, * C_device;
    hipMalloc((void**)&A_device, N * K * sizeof(double));
	hipMalloc((void**)&B_device, M * K * sizeof(double));
	hipMalloc((void**)&C_device, N * M * sizeof(double));
	hipMemcpy(A_device, A, N * K * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(B_device, B, M * K * sizeof(double), hipMemcpyHostToDevice);

    clock_t t;
	t = clock();

	dim3 threads(16, 16, 1), grid(divUp(M, 16), divUp(N, 16), 1);
	mat_prod <<< grid, threads >>> (A_device, B_device, C_device, N, K, M);
	hipDeviceSynchronize();
	
	t = clock() - t;

	hipMemcpy(C, C_device, N * M* sizeof(double), hipMemcpyDeviceToHost);

	hipFree(A_device);
	hipFree(B_device);
	hipFree(C_device);
	free(A);
	free(B);
	free(C);

	printf("%f", ((double)t) / CLOCKS_PER_SEC);
    return 0;
}