#include "hip/hip_runtime.h"
﻿#include <stdio.h>

#include <hip/hip_runtime.h>
#include <>

#include <opencv2/highgui/highgui.hpp>

using namespace cv;

__global__ void AlphaMatting(uchar3* image_source_1, uchar3* image_source_2, uchar* image_mask, uchar3* image_result, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N) {
		int alpha = (image_mask[idx] > 255 / 2) ? 1 : 0;
		image_result[idx].x = image_source_1[idx].x * alpha + (1 - alpha) * image_source_2[idx].x;
		image_result[idx].y = image_source_1[idx].y * alpha + (1 - alpha) * image_source_2[idx].y;
		image_result[idx].z = image_source_1[idx].z * alpha + (1 - alpha) * image_source_2[idx].z;
	}
}

int divUp(int a, int b) {
	return (a + b - 1) / b;
}

int main() {
	
	hipSetDevice(0);
	
	Mat frame_source_1 = imread("./greenScreen.jpg", IMREAD_COLOR),
		frame_source_2 = imread("./fondo.bmp", IMREAD_COLOR),
		frame_mask = imread("./greenscreenMask.bmp", IMREAD_GRAYSCALE),
		frame_result;

	int imageW = frame_source_1.cols, imageH = frame_source_1.rows, N = imageW * imageH;

	uchar3* image_source_1, * image_source_2, * image_result;
	uchar* image_mask;
	hipMalloc((void**)&image_source_1, N * sizeof(uchar3));
	hipMalloc((void**)&image_source_2, N * sizeof(uchar3));
	hipMalloc((void**)&image_result, N * sizeof(uchar3));
	hipMalloc((void**)&image_mask, N * sizeof(uchar));
	hipMemcpy(image_source_1, frame_source_1.data, N * sizeof(uchar3), hipMemcpyHostToDevice);
	hipMemcpy(image_source_2, frame_source_2.data, N * sizeof(uchar3), hipMemcpyHostToDevice);
	hipMemcpy(image_mask, frame_mask.data, N * sizeof(uchar), hipMemcpyHostToDevice);

	int threads = 512, grid = divUp(N, threads);

	AlphaMatting <<< grid, threads >>> (image_source_1, image_source_2, image_mask, image_result, N);
	hipDeviceSynchronize();

	frame_result.create(imageH, imageW, CV_8UC(3));
	hipMemcpy(frame_result.data, image_result, N * sizeof(uchar3), hipMemcpyDeviceToHost);
	imshow("ImagenResultado", frame_result);

	waitKey(0);

	frame_result.release();

	hipFree(image_source_1);
	hipFree(image_source_2);
	hipFree(image_mask);
	hipFree(image_result);

	return 0;
}