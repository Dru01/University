#include "hip/hip_runtime.h"
﻿#include <stdio.h>

#include <hip/hip_runtime.h>
#include <>

#include <opencv2/highgui/highgui.hpp>

using namespace cv;

__constant__ int k[2][3][3] = {
	{
		{-1, 0, 1},
		{-2, 0, 2},
		{-1, 0, 1}
	},
	{
		{-1, -2, -1},
		{0, 0, 0},
		{1, 2, 1}
	}
};

__global__ void Derivates(uchar* image, int* Dx, int* Dy, int imageW, int imageH, int N) 
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int index, index1;

	if (0 < idx && idx < imageW - 1 && 0 < idy && idy < imageH - 1)
	{
		index = (idy) * imageW + (idx);
		Dx[index] = 0;
		Dy[index] = 0;
		for (int k_i = -1; k_i < 2; k_i++)
		{
			for (int k_j = -1; k_j < 2; k_j++)
			{
				index1 = (idy + k_i) * imageW + (idx + k_j);
				Dx[index] += image[index1] * k[0][k_i + 1][k_j + 1];
				Dy[index] += image[index1] * k[1][k_i + 1][k_j + 1];
			}
		}
	}
}

__global__ void BordersinImage(int* Dx, int* Dy, uchar* MG, uchar* MGT, int imageW, int imageH, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int index = (idy)*imageW + (idx);

	if (0 < idx && idx < imageW - 1 && 0 < idy < imageH - 1)
		MG[index] = fminf(sqrtf(Dx[index] * Dx[index] + Dy[index] * Dy[index]), 255);
	else if (0 <= idx && idx < imageW && 0 <= idy && idy < imageH)
		MG[index] = 0;
	MGT[index] = (MG[index] > 255 / 2 ? 255 : 0);
}

int divUp(int a, int b) {
	return (a + b - 1) / b;
}

int main() {
	
	hipSetDevice(0);
	
	Mat frame = imread("./pinzas_gray.png", IMREAD_GRAYSCALE), frameMG, frameMGT;

	int imageW = frame.cols, imageH = frame.rows, N = imageW * imageH;

	// Crear memoria en la GPU
	uchar* image, * MG, * MGT;
	int* Dx, * Dy;
	hipMalloc((void**)&image, N * sizeof(uchar));
	hipMalloc((void**)&Dx, N * sizeof(int));
	hipMalloc((void**)&Dy, N * sizeof(int));
	hipMalloc((void**)&MG, N * sizeof(uchar));
	hipMalloc((void**)&MGT, N * sizeof(uchar));

	hipMemcpy(image, frame.data, N * sizeof(uchar), hipMemcpyHostToDevice);

	dim3 threads(16, 16, 1), grid(divUp(imageW, 16), divUp(imageH, 16), 1);

	Derivates <<< grid, threads >>> (image, Dx, Dy, imageW, imageH, N);
	hipDeviceSynchronize();
	BordersinImage <<< grid, threads >>> (Dx, Dy, MG, MGT, imageW, imageH, N);
	hipDeviceSynchronize();

	frameMG.create(imageH, imageW, CV_8UC(1));
	frameMGT.create(imageH, imageW, CV_8UC(1));
	hipMemcpy(frameMG.data, MG, N * sizeof(uchar), hipMemcpyDeviceToHost);
	hipMemcpy(frameMGT.data, MGT, N * sizeof(uchar), hipMemcpyDeviceToHost);
	imshow("MG", frameMG);
	imshow("MGT", frameMGT);
	waitKey(0);

	frameMG.release();
	frameMGT.release();

	hipFree(image);
	hipFree(Dx);
	hipFree(Dy);
	hipFree(MG);
	hipFree(MGT);

	return 0;
}